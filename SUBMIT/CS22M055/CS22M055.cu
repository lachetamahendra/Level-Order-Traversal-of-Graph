#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;


ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/


//Kernel to calculate the end of node of a give level in a graph.
__global__ void find_level_start_end(int* d_offset, int* d_csrList, int s, int e, int *d_max_node) {
  int id = s + blockIdx.x * blockDim.x + threadIdx.x;
  if (id <= e) {
      if(d_csrList[d_offset[id + 1] - 1] > *d_max_node)
          atomicMax(&d_max_node[0] , d_csrList[d_offset[id + 1] - 1]);
          
  } 
}

//kernel to calculate the active indegree of each level except level 0.

__global__ void find_active_indegree(int* d_an, int* d_aid, int s, int e, int* d_offset, int* d_csrList) {
  int id = s + blockIdx.x * blockDim.x + threadIdx.x;
  if (id <= e) {
    if (d_an[id] == 1) {
        for (int i = d_offset[id]; i < d_offset[id + 1]; i++) {
            atomicAdd(&d_aid[d_csrList[i]],1);
        }
    }
  }
}
     
//kernel to calculate the active node for each level.  
__global__ void find_active_node(int *d_an,int *d_aid,int *d_apr, int s, int e) {
  int id =s + blockIdx.x * blockDim.x + threadIdx.x;
  if (id <= e) {
      if (d_aid[id] >= d_apr[id] ) {
          d_an[id] = 1;
      }
  }
}

//find deactive node from following active node.
__global__ void  find_deactivate_node(int *d_an, int s, int e ){
  int id =s + blockIdx.x * blockDim.x + threadIdx.x + 1;
  if (id < e) {
      if (d_an[id] == 1 && d_an[id - 1] == 0 && d_an[id + 1] == 0) {
          d_an[id] = 0;
      }
  }
}

//kernel to calculate the number of active nodes at each level.
__global__ void active_node_count(int *d_activeVertex, int *d_an, int s, int e, int level) {
  int id = s + blockIdx.x * blockDim.x + threadIdx.x;
  if (id <= e) {
      if (d_an[id] == 1) {
          atomicAdd(&d_activeVertex[level],1);
      }
  }
}
    
    
    
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    // int *d_activeVertex;
	// hipMalloc(&d_activeVertex, L*sizeof(int));


/***Important***/

// Initialize d_aid array to zero for each vertex
// Make sure to use comments

/***END***/
double starttime = rtclock(); 

/*********************************CODE AREA*****************************************/



  int i = 0;
  int *h_level;  //It use to represent the starting and ending index of level.
  int *d_max_node,*h_max_node;//it use to store the last node of given level.
  h_level = (int*)malloc(L * 2 * sizeof(int));
  memset(h_level, 0 , L * 2 * sizeof(int));
  hipMalloc(&d_max_node, sizeof(int));
  h_max_node = (int*)malloc(sizeof(int));
  h_level[0] = 0;
  int j = 0;
  while (h_apr[j] == 0) {
      j++;
  }
  h_level[1] = j - 1;
  //kernel call for level calcuation.
  for (int i = 0; i < 2 * (L - 1); i = i + 2) {
      hipMemset(d_max_node, 0, sizeof(int));
      //kerenl call.
      int gridl = ceil((float)(h_level[i+1] - h_level[i] + 1) / 1024);
      dim3 grid(gridl, 1, 1);
      find_level_start_end << < grid, 1024 >> > (d_offset,d_csrList,h_level[i],h_level[i+1],d_max_node);

      hipMemcpy(h_max_node, d_max_node, sizeof(int), hipMemcpyDeviceToHost);
      h_level[i+2] = h_level[i+1]+1;
      h_level[i+3] = *h_max_node;
  }



  int *h_active_node,*d_an;//Index represent the node and value represent the active node if 1 or inactive node if 0.
  int *h_active_indegree;//Store the active indegree of node.

  h_active_node = (int*)malloc(V * sizeof(int));
  h_active_indegree = (int*)malloc(V * sizeof(int));

  hipMalloc(&d_an, sizeof(int) * V);

  memset(h_active_node, 0, V * sizeof(int));
  memset(h_active_indegree, 0, V * sizeof(int));
  
  //first level all nodes are active.
  i = 0;
  while (i <= h_level[1]) {
      h_active_node[i] = 1;
      i++;
  }

  hipMemcpy(d_an, h_active_node, sizeof(int) * V, hipMemcpyHostToDevice);
  hipMemcpy(d_aid, h_active_indegree, sizeof(int) * V, hipMemcpyHostToDevice);
  i = 0;
  while (i < 2*(L - 1)) {

      //Kernel call to find the active in degree of nodes in level i.
      int gridx=ceil((float)(h_level[i + 1] - h_level[i] + 1) / 1024);
      dim3 grid1(gridx,1,1);
      find_active_indegree << < grid1 , 1024 >> > (d_an, d_aid, h_level[i], h_level[i + 1], d_offset, d_csrList);
      hipDeviceSynchronize();

      //kernel call to find the active node in level i+1.
      gridx=ceil((float)(h_level[i + 3] - h_level[i + 2] + 1) / 1024);
      dim3 grid2(gridx,1,1);
      find_active_node << < grid2 , 1024 >> > (d_an, d_aid, d_apr, h_level[i + 2], h_level[i + 3]);
      hipDeviceSynchronize();

      //kernel call to find dective node in level i+1
      gridx=ceil((float)(h_level[i + 3] - h_level[i + 2] + 1) / 1024);
      dim3 grid3(gridx,1,1);
      find_deactivate_node<<<grid3,1024>>>(d_an, h_level[i + 2], h_level[i + 3]);
      hipDeviceSynchronize();
      i = i + 2;
  }
  
  //compute count of active node at each level.
  int* d_activeVertex;
  hipMalloc(&d_activeVertex, L*sizeof(int));
  hipMemset(d_activeVertex, 0, L * sizeof(int));
  for (int i = 0; i < 2*L; i=i+2) {
      //kernel calling.
      int gridx=ceil((float)(h_level[i + 1] - h_level[i] + 1) / 1024);
      dim3 grid3(gridx,1,1);
      active_node_count << < grid3 , 1024 >> > (d_activeVertex, d_an, h_level[i], h_level[i + 1],i/2);
  }
  hipMemcpy(h_activeVertex, d_activeVertex, sizeof(int) * L, hipMemcpyDeviceToHost);

  //deallocate cudamemory
  hipFree(d_an);
  hipFree(d_max_node);

  //deallocate memory
  free(h_level);
  free(h_max_node);
  free(h_active_indegree);
  free(h_active_node);

/********************************END OF CODE AREA**********************************/
double endtime = rtclock();  
printtime("GPU Kernel time: ", starttime, endtime);  

// --> Copy C from Device to Host
char outFIle[30] = "./output.txt" ;
printResult(h_activeVertex, L, outFIle);
if(argc>2)
{
    for(int i=0; i<L; i++)
    {
        printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
    }
}

    return 0;
}
